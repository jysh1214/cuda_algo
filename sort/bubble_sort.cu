#include <stdio.h>
#include <stdlib.h>
#include <time.h>
#include <assert.h>
#include <hip/hip_runtime.h>
#include <algorithm>

#define u32 unsigned int
#define u64 unsigned long
#define uchar unsigned char
#define BLOCK_SIZE 64
#define FULL_MASK 0xffffffff

/** !ISSUE
 * sometimes sorted array is partially change, why?
 * e.g.
 * CPU SORT: 1, 2, 3, 3, 4, 5
 * GPU SORT: 1, 2, 2, 3, 4, 5
 */

#define CREATE_RAND_ARR(arr, size, min, max) \
do {                                         \
    time_t t;                                \
    srand((unsigned)time(&t));               \
    for (u32 i = 0; i < size; i++)           \
        arr[i] = rand() % max + min;         \
} while (0)                                  \

#define COPY_ARR(a, b, size)       \
do {                               \
    for (u32 i = 0; i < size; i++) \
        a[i] = b[i];               \
} while (0)                        \

#define PRINT_ARR(arr, size)       \
do {                               \
    for (u32 i = 0; i < size; i++) \
        printf("%u, ", arr[i]);    \
    printf("\n");                  \
} while (0)                        \

#define ARR_EQ(a, b, size)           \
do {                                 \
    bool eq = true;                  \
    for (u32 i = 0; i < size; i++) { \
        if (a[i] != b[i]) {          \
            eq = false;              \
            break;                   \
        }                            \
    }                                \
    if (eq)                          \
        printf("EQUAL\n");           \
    else                             \
        printf("NOT EQAUL\n");       \
} while (0)                          \

void check(const u32* a, const u32 size)
{
    bool correct = true;
    for (u32 i = 0; i < size - 1; i++) {
        if (a[i] > a[i + 1]) {
            printf("%u, %u\n", a[i] , a[i + 1]);
            correct = false;
            break;
        }
    }

    if (correct) {
        printf("CORRECT\n");
    }
    else {
        printf("NOT CORRECT\n");
    }
}

int compare(const void* a, const void* b)
{
    int c = *(int*)a;
    int d = *(int*)b;
    if(c < d) {return -1;}
    else if (c == d) {return 0;}
    else return 1;
}

__global__
void global_bubble_sort_kernel(const u32* __restrict__ arr, const u32 size, u32* sorted_arr)
{
    u32 id_x = blockIdx.x * blockDim.x + threadIdx.x;
    if (id_x >= size) {
        return;
    }

    sorted_arr[id_x] = arr[id_x];
    __syncthreads();
    int key = 0;
    for (u32 n = size * 2; n > 0; n--) {
        if ((id_x & 1) == key && (id_x < size - 1)) {
            if (sorted_arr[id_x] > sorted_arr[id_x + 1]) {
                // sorted_arr[id_x] = sorted_arr[id_x + 1];
                // memcpy(sorted_arr + id_x + 1, &temp, sizeof(u32));
                // atomicExch(&sorted_arr[id_x], sorted_arr[id_x + 1]);
                u32 temp1 = sorted_arr[id_x];
                u32 temp2 = sorted_arr[id_x + 1];
                atomicCAS(&sorted_arr[id_x], temp1, sorted_arr[id_x + 1]);
                atomicCAS(&sorted_arr[id_x + 1], temp2, temp1);
            }
        }
        key ^= 1;
        __syncthreads();
    }
}

void gpuGlobalBubbleSort(const u32* h_arr, const u32 size, u32* h_sorted_arr)
{
    assert(size > 0);

    u32* d_arr;
    hipMalloc((void**)&d_arr, size * sizeof(u32));
    hipMemcpy(d_arr, h_arr, size * sizeof(u32), hipMemcpyHostToDevice);

    u32* d_sorted_arr;
    hipMalloc((void**)&d_sorted_arr, size * sizeof(u32));

    dim3 blocks = BLOCK_SIZE;
    dim3 grids = (size + BLOCK_SIZE - 1) / BLOCK_SIZE;
    global_bubble_sort_kernel <<< grids, blocks >>> (d_arr, size, d_sorted_arr);
    if (hipSuccess != hipGetLastError()) {
        printf("global_bubble_sort_kernel fault!\n");
    }

    hipMemcpy(h_sorted_arr, d_sorted_arr, size * sizeof(u32), hipMemcpyDeviceToHost);

    hipFree(d_arr);
    hipFree(d_sorted_arr);
}

/**
 * @ref https://github.com/Kshitij421/Odd-Even-Sort-using-Cuda-/blob/master/oddeven.cu
 * @ref https://github.com/znahman/parallel-comparison-bubblesort/blob/master/parallel-bubsort.cu
 */
int main()
{
    const u32 size = 10000;
    u32* arr = (u32*)malloc(size * sizeof(u32));
    CREATE_RAND_ARR(arr, size, 0, 10000);

    // PRINT_ARR(arr, size);
    
    u32* cpuSortArr = (u32*)malloc(size * sizeof(u32));
    COPY_ARR(cpuSortArr, arr, size);
    qsort((void*)cpuSortArr, size, sizeof(u32), compare);
    // PRINT_ARR(cpuSortArr, size);

    u32* gpuGlobalBubbleSortArr = (u32*)malloc(size * sizeof(u32));
    gpuGlobalBubbleSort(arr, size, gpuGlobalBubbleSortArr);
    // PRINT_ARR(gpuGlobalBubbleSortArr, size);
    check(gpuGlobalBubbleSortArr, size);

    // ARR_EQ(gpuGlobalBubbleSortArr, cpuSortArr, size);

    free(arr);
    free(cpuSortArr);
    free(gpuGlobalBubbleSortArr);
    
    return 0;
}